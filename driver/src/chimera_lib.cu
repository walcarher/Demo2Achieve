#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <fcntl.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "../altera_dma_cmd.h"

// Global variables
ssize_t f;
char *buf;
struct dma_cmd cmd;
int *tensor;
int *dev_ptr;
// Max number of threads per block on JetsonTX2
// On-Chip memory of 64Kb - 2048 DWORDS (32b)
const dim3 threads(1024);
const dim3 blocks(2048/threads.x);

int init_tensor(int length, int *tensor)
{
	int i = 0;
	for (i = 0; i < length; i++) {
		tensor[i] = 0;
	}
	return 0;
}

int print_tensor(int length, int *tensor)
{
	int i = 0;
	for (i = 0; i < length; i++) {
		printf("Tensor value = %d\n", tensor[i]);
	}
	return 0;	
}

int write_to_fpga_raw(int *tensor)
{
	// Reads Tensor from CPU/GPU to FPGA
	ioctl(f, ALTERA_IOCX_READ_TENSOR, tensor);
	ioctl(f, ALTERA_IOCX_WAIT);
	return 0;
}

int * read_from_fpga_raw(int *tensor)
{
	// Writes Tensor from FPGA to CPU/GPU
	ioctl(f, ALTERA_IOCX_WRITE_TENSOR, tensor);
	ioctl(f, ALTERA_IOCX_WAIT);
	return tensor;
}

int read_status(){
	cmd.cmd = ALTERA_CMD_READ_STATUS;
	cmd.buf = buf;
	write (f, &cmd, 0);
	return 0;
}

__global__ void read_from_gpu(torch::PackedTensorAccessor32<int, 1> accessor, int* tensor_ptr,int length, int offset){
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	tensor_ptr[tid] = accessor[tid+offset*length];
}

int open_fpga()
{
	f = open ("/dev/altera_dma", O_RDWR);
	if (f == -1) {
        printf ("Error: Couldn't open the FPGA device.\n");
        return 1;
    } else {
        printf ("FPGA device successfully opened: file handle #%lu!\n", (long unsigned int)f);
		struct dma_cmd cmd;
		buf = (char*)malloc(sizeof(struct dma_status));
		cmd.cmd = ALTERA_CMD_READ_STATUS;
        cmd.buf = buf;
        write (f, &cmd, 0);
		hipHostAlloc((void **)&tensor, ((struct dma_status *)buf)->altera_dma_num_dwords*sizeof(int), hipHostMallocMapped);
		hipHostGetDevicePointer((void **)&dev_ptr, (void *)tensor, 0);
		init_tensor(((struct dma_status *)buf)->altera_dma_num_dwords, tensor);
		return 0;
	}
}

int close_fpga()
{
	free(buf);
	hipHostFree(tensor);
	hipFree(dev_ptr);
	hipDeviceReset();
	close (f);
	return 0;
}

int write_to_fpga(torch::Tensor torch_tensor)
{
	if (torch_tensor.dim() != 4){
		printf("Error: Only 2D-Conv supported. Tensor must be dimension 4.\n");
		return 1;
	}
	if (torch_tensor.device().is_cpu()) {
		auto tensor_acc = torch_tensor.accessor<int, 4>();
		int C = tensor_acc.size(1);
		int H = tensor_acc.size(2);
		int W = tensor_acc.size(3);
		int c, h, w;
		int last_c = 0, last_h = 0, last_w = 0;
		int addr_c = 0, addr_h = 0, addr_w = 0;
		
		for (h = 0 ; h < H; h++) {
			for (w = 0 ; w < W; w++) {
				for (c = 0 ; c < C; c++) {
					if (addr_c+addr_w*C+addr_h*C*W == ((struct dma_status *)buf)->altera_dma_num_dwords-1){
						last_c = c;
						last_h = h;
						last_w = w;
						addr_c = c - last_c;
						addr_h = h - last_h;
						addr_w = w - last_w;
						write_to_fpga_raw(tensor);
					} else {
						addr_c = c - last_c;
						addr_h = h - last_h;
						addr_w = w - last_w;
					}
					// Index unrolling
					tensor[addr_c+addr_w*C+addr_h*C*W] = tensor_acc[0][c][h][w];
				}
			}
		}
		write_to_fpga_raw(tensor);
	} else {
		int C = torch_tensor.size(1);
		int H = torch_tensor.size(2);
		int W = torch_tensor.size(3);
		// Flatten tensor with with priority on the number of channels (C) or tensor depth
		torch::Tensor temp_tensor = torch_tensor.permute({0,3,2,1}).permute({0,2,1,3}).reshape({C*H*W});
		auto tensor_acc = temp_tensor.packed_accessor32<int, 1>();
		int chunks_num = (int)ceil(C*H*W / ((struct dma_status *)buf)->altera_dma_num_dwords);
		for (int i = 0; i < chunks_num; i++){
			// Maximum number of threads per block (1024) on the TX2 Pascal arch
			// Split tensor into accesses of N blocks with 1024 threads
			read_from_gpu<<<blocks,threads>>>(tensor_acc, dev_ptr, ((struct dma_status *)buf)->altera_dma_num_dwords, i);
			//print_tensor(((struct dma_status *)buf)->altera_dma_num_dwords, tensor);
			hipDeviceSynchronize();
			write_to_fpga_raw(tensor); 
		}
	}
	return 0;
}

torch::Tensor read_from_fpga(torch::Tensor torch_tensor)
{
	if (torch_tensor.dim() != 4){
		printf("Error: Only 2D-Conv supported. Tensor must be dimension 4.\n");
		return torch_tensor;
	}
	tensor = read_from_fpga_raw(tensor);
	//print_tensor(((struct dma_status *)buf)->altera_dma_num_dwords, tensor);
	torch_tensor = torch::from_blob(tensor, {torch_tensor.size(0),torch_tensor.size(1),torch_tensor.size(2),torch_tensor.size(3)}, torch::dtype(torch::kInt32));
	return torch_tensor;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
	m.def("open", &open_fpga, "Open FPGA device");
	m.def("close", &close_fpga, "Close FPGA device");
	m.def("write", &write_to_fpga, "Tensor write to FPGA");
	m.def("read", &read_from_fpga, "Tensor read from FPGA");
}